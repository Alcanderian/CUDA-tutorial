#include <stdio.h>
#include <math.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <mkl_cblas.h>

#include "interfaces.h"
#include "kernels.cuh"
#include "kernels.h"
#include "../prof.h"

void gpu_sgemm(
    float *a, float *b, float *c,
    size_t N, size_t M, size_t K,
    float alpha, float beta, int kernel_type)
{
    float *dev_a = 0;
    float *dev_b = 0;
    float *dev_c = 0;

    hipblasHandle_t handle;

    hs_timer timer;
    timer.tic("gpu sgemm");

    if (kernel_type == 'b')
        hipblasCreate(&handle);

    hipMalloc((void **)&dev_a, M * K * sizeof(float));
    hipMalloc((void **)&dev_b, K * N * sizeof(float));
    hipMalloc((void **)&dev_c, M * N * sizeof(float));

    hipMemcpy(dev_a, a, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, K * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, M * N * sizeof(float), hipMemcpyHostToDevice);

    switch (kernel_type)
    {
    case 0:
    {
        int grid_r = M / 32;
        int grid_c = N / 32;
        if (M % 32 != 0)
            grid_r += 1;
        if (N % 32 != 0)
            grid_c += 1;
        dim3 grid_d(grid_r, grid_c, 1);
        dim3 block_d(32, 32, 1);
        cuda_kernel_sgemm_0<<<grid_d, block_d>>>(dev_a, dev_b, dev_c, N, M, K, alpha, beta);
        break;
    }
    case 1:
    {
        int grid_r = M / 32;
        int grid_c = N / 32;
        if (M % 32 != 0)
            grid_r += 1;
        if (N % 32 != 0)
            grid_c += 1;
        dim3 grid_d(grid_r, grid_c, 1);
        dim3 block_d(32, 32, 1);
        cuda_kernel_sgemm_1<<<grid_d, block_d>>>(dev_a, dev_b, dev_c, N, M, K, alpha, beta);
        break;
    }
    case 'b':
    {
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, dev_b, N, dev_a, K, &beta, dev_c, N);
        break;
    }
    }

    hipDeviceSynchronize();

    hipMemcpy(c, dev_c, M * N * sizeof(float), hipMemcpyDeviceToHost);

    if (kernel_type == 'b')
        hipblasDestroy(handle);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    timer.toc("gpu sgemm");
}

void gpu_warmup()
{
    float *dev_p = 0;

    hs_timer timer;
    timer.tic("gpu warmup");

    hipMalloc((void **)&dev_p, 16 * 32 * sizeof(float));

    cuda_kernel_warmup<<<16, 32>>>(dev_p);

    hipDeviceSynchronize();

    hipFree(dev_p);

    timer.toc("gpu warmup");
}

void cpu_sgemm(
    float *a, float *b, float *c,
    size_t N, size_t M, size_t K,
    float alpha, float beta, int kernel_type)
{
    hs_timer timer;
    timer.tic("cpu sgemm");

    switch (kernel_type)
    {
    case 0:
    {
        cpu_kernel_sgemm_0(a, b, c, N, M, K, alpha, beta);
        break;
    }
    case 'm':
    {
        cblas_sgemm(CblasColMajor, CblasNoTrans, CblasNoTrans, N, M, K, alpha, b, N, a, K, beta, c, N);
        break;
    }
    }
    timer.toc("cpu sgemm");
}

void cpu_warmup()
{
    hs_timer timer;
    timer.tic("cpu warmup");

    const size_t arr_size = 1024;
    float *p = new float[arr_size];

#pragma omp parallel for simd
    for (size_t i = 0; i < arr_size; i++)
    {
        float f = (float)i;
        p[i] = f * f * f;
    }

    delete p;

    timer.toc("cpu warmup");
}
